#include "hip/hip_runtime.h"
/*
 * Author: Evandro C Taquary
 * Compilation: nvcc -arch=sm_35 gpu_replication.cu modcpy.cu -o gpu
 * 
 * */

#include <iostream>
#include <sys/time.h>
#include "modcpy.h"

using namespace std;

#define CHECK(call) \
{ \
        const hipError_t error = call; \
        if (error != hipSuccess) \
        { \
                cout << "Error: " << __FILE__ ": " << __LINE__ << ", "; \
                cout << "code: "<< error << ", reason: " << hipGetErrorString(error) << endl; \
                exit(EXIT_FAILURE); \
        } \
}

typedef struct {
	short parent;
	short child1;
	short child2;
	double branch;
}node;

int main(int argc, char *argv[])
{	
	if(argc != 3){
		cout << "Usage: " << argv[0] << " #nodes #replications" << endl;
		exit(EXIT_FAILURE);
	}
	
	const int NODS = atoi(argv[1]);
	const int REPS = atoi(argv[2]);
	
	struct timeval begin, end;
	double time_spent;	
	
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device,0);
	
	node tree[NODS];
	node *d_tree;

	size_t treeBytes = sizeof(tree);
	size_t repBytes = treeBytes * REPS;
	
	node *h_replics = (node*) malloc(repBytes);
	node *d_replic;

	int blockSize = device.warpSize*32;
	int gridSize = ((repBytes/sizeof(int4))/blockSize);
	dim3 grid = dim3(gridSize);
	dim3 block = dim3(blockSize);

	for(int i=0; i<NODS; i++)
	{
		tree[i].parent = i;
		tree[i].child1 = i;
		tree[i].child2 = i;
		tree[i].branch = i;
	}
	
	CHECK(hipMalloc((void **) &d_tree, treeBytes));
	CHECK(hipMemcpy(d_tree, &tree, treeBytes, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void **) &d_replic, repBytes));
	
/**********************PARALLEL MEASUREMENT**********************/
	gettimeofday(&begin, NULL);
	modcpy<<<grid, block>>>(d_replic, d_tree, repBytes, treeBytes);
	CHECK(hipDeviceSynchronize());
	gettimeofday(&end, NULL);
	time_spent = (double) (end.tv_usec - begin.tv_usec)/1000 + (end.tv_sec - begin.tv_sec)*1000;
	cout << "Time spent:\t" << time_spent << "ms " <<  endl;
/**********************PARALLEL MEASUREMENT**********************/		

	CHECK(hipMemcpy(h_replics, d_replic, repBytes, hipMemcpyDeviceToHost));

	for(int i=0; i<REPS; i++)
		for(int j=0; j<NODS; j++)		
			if(	h_replics[i*NODS+j].parent != j ||
				h_replics[i*NODS+j].child1 != j ||
				h_replics[i*NODS+j].child2 != j ||
				h_replics[i*NODS+j].branch != j )
				{ 	
					cout << "Data doesn't match!" << endl;
					exit(1); 
				}
	cout << "Data does match!" << endl;
	hipDeviceReset();	
  exit(EXIT_SUCCESS);
}
