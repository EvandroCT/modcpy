#include "hip/hip_runtime.h"
/*************************************************************************
	
	Copyright (C) 2016	Evandro Taquary
	
	This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
	
*************************************************************************/

#include <iostream>
#include <sys/time.h>
#include "modcpy.h"

using namespace std;

#define CHECK(call) \
{ \
        const hipError_t error = call; \
        if (error != hipSuccess) \
        { \
                cout << "Error: " << __FILE__ ": " << __LINE__ << ", "; \
                cout << "code: "<< error << ", reason: " << hipGetErrorString(error) << endl; \
                exit(EXIT_FAILURE); \
        } \
}

int main(int argc, char *argv[])
{
	
	hipSetDevice(0);
	if(argc != 3){
		cout << "Usage: " << argv[0] << " #nodes #replications" << endl;
		exit(EXIT_FAILURE);
	}
	
	const int NODS = atoi(argv[1]);
	const int REPS = atoi(argv[2]);	
	
	size_t treeBytes = 3*sizeof(short)*NODS + sizeof(double)*NODS;
	int r = treeBytes%sizeof(int4);	
	treeBytes += r ? sizeof(int4)-r : 0;		
	size_t repBytes = treeBytes * REPS;
	
	void *d_tree;
	void *h_tree = (void*) malloc (treeBytes);
	memset(h_tree,0,treeBytes);
	short *parent = (short*) h_tree;
	short *child1 = parent+NODS;
	short *child2 = child1+NODS;
	double *branch = (double*) (child2+NODS);	
	
	for(short i=0; i<NODS; i++){
		parent[i] = i;
		child1[i] = i;
		child2[i] = i;
		branch[i] = i;
	}	
	
	void *h_replics = (void*) malloc(repBytes);
	void *d_replics;

	hipDeviceProp_t device;
	hipGetDeviceProperties(&device,0);
	
	int blockSize = device.warpSize*32;
	int gridSize = (repBytes/sizeof(int4) + (blockSize-1)) / blockSize;
	dim3 grid = dim3(gridSize);
	dim3 block = dim3(blockSize);

	CHECK(hipMalloc((void**) &d_tree, treeBytes));
	CHECK(hipMemcpy(d_tree, h_tree, treeBytes, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void **) &d_replics, repBytes));

	struct timeval begin, end;
	float time_spent;
	
/*******************************PARALLEL MEASUREMENT*******************************/
	
	gettimeofday(&begin, NULL);
	modcpy<<<grid, block>>>(d_replics, d_tree, repBytes, treeBytes);
	CHECK(hipDeviceSynchronize());
	gettimeofday(&end, NULL);
	time_spent = (float) (end.tv_usec - begin.tv_usec)/1000 + (end.tv_sec - begin.tv_sec)*1000;
	cout << "Time spent:\t" << time_spent << "ms " <<  endl;
	
/*******************************PARALLEL MEASUREMENT*******************************/		

	CHECK(hipMemcpy(h_replics, d_replics, repBytes, hipMemcpyDeviceToHost));

	for(int i=0; i<REPS; i++){
		parent = (short*) (h_replics+treeBytes*i);
		child1 = parent+NODS;
		child2 = child1+NODS;
		branch = (double*) (child2+NODS);		
		for(int j=0; j<NODS; j++)
		{
			if(parent[j] != j || child1[j] != j || child2[j] != j || branch[j] != j ){
				cout << "Data doesn't match!" << endl;
				exit(EXIT_FAILURE);
			}
		}
	}	
	cout << "Data does match!" << endl;
	hipDeviceReset();	
	exit(EXIT_SUCCESS);
}
