/*
 * Author: Evandro C Taquary
 * Compilation: nvcc -arch=sm_35 cpu_transfer.cu -o cpu
 * 
 * */


#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

#define CHECK(call) \
{ \
        const hipError_t error = call; \
        if (error != hipSuccess) \
        { \
                cout << "Error: " << __FILE__ ": " << __LINE__ << ", "; \
                cout << "code: "<< error << ", reason: " << hipGetErrorString(error) << endl; \
                exit(EXIT_FAILURE); \
        } \
}

typedef struct {
	short parent;
	short child1;
	short child2;
	double branch;
}node;

int main(int argc, char *argv[])
{	
	if(argc != 3){
		cout << "Usage: " << argv[0] << " #nodes #replications" << endl;
		exit(EXIT_FAILURE);
	}
	
	const int NODS = atoi(argv[1]);
	const int REPS = atoi(argv[2]);
	
	struct timeval begin, end;
	double time_spent;
	
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device,0);	

	size_t repBytes = NODS * REPS * sizeof(node);
	
	node *h_replics = (node*) malloc(repBytes);
	node *d_replics;

	for(int i=0; i<REPS; i++)
		for(int j=0; j<NODS; j++)
		{
			h_replics[i*NODS+j].parent = j;
			h_replics[i*NODS+j].child1 = j;
			h_replics[i*NODS+j].child2 = j;
			h_replics[i*NODS+j].branch = j;
		}
	
	CHECK(hipMalloc((void **) &d_replics, repBytes));
	
/**********************SERIAL MEASUREMENT**********************/
	gettimeofday(&begin, NULL);
	CHECK(hipMemcpy(d_replics, h_replics, repBytes, hipMemcpyHostToDevice));
	gettimeofday(&end, NULL);
	time_spent = (double) (end.tv_usec-begin.tv_usec)/1000 + (end.tv_sec-begin.tv_sec)*1000;
	cout << "Time spent:\t" << time_spent << "ms " <<  endl;
/**********************SERIAL MEASUREMENT**********************/
	
	for(int i=0; i<REPS; i++)
		for(int j=0; j<NODS; j++)
		{
			h_replics[i*NODS+j].parent = 0;
			h_replics[i*NODS+j].child1 = 0;
			h_replics[i*NODS+j].child2 = 0;
			h_replics[i*NODS+j].branch = 0;
		}
		
	CHECK(hipMemcpy(h_replics, d_replics, repBytes, hipMemcpyDeviceToHost));

	for(int i=0; i<REPS; i++)
		for(int j=0; j<NODS; j++)		
			if(	h_replics[i*NODS+j].parent != j ||
				h_replics[i*NODS+j].child1 != j ||
				h_replics[i*NODS+j].child2 != j ||
				h_replics[i*NODS+j].branch != j )
				{ 	
					cout << "Data doesn't match!" << endl;
					exit(1); 
				}
	cout << "Data does match!" << endl;
	hipDeviceReset();	
  exit(EXIT_SUCCESS);
}
